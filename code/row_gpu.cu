#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define uint32_t unsigned int
#define MAX_THREADS 512
#define MAX_PATTERN_SIZE 1024
#define MAX_BLOCKS 8
#define MAX_STREAMS 16
#define TEXT_MAX_LENGTH 1000000000
void calculateBBArray(uint32_t** BB,const char* pattern_h,int m,int k , int lc , int J);

void checkCUDAError(const char *msg)
{
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) 
        {   
                fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                hipGetErrorString( err) );
                exit(EXIT_FAILURE);
        }    
}
char* getTextString()
{
   FILE *input, *output;
   char c;
   char * inputbuffer=(char *)malloc(sizeof(char)*TEXT_MAX_LENGTH);
   
   int numchars = 0, index  = 0;

   input = fopen("sequence.fasta", "r");
   c = fgetc(input);
   while(c != EOF)
   {
	inputbuffer[numchars] = c;
  	numchars++;
	c = fgetc(input);
   }
   fclose(input);
   inputbuffer[numchars] = '\0'; 
   return inputbuffer;
}


__global__ void match(uint32_t* BB_d,const char* text_d,int n, int m,int k,int J,int lc,int start_addr,int textBlockSize,int overlap ,int* matched)
{
	__shared__ int D[MAX_THREADS+2];
	__shared__ char Text_S[MAX_PATTERN_SIZE];
	__shared__ int DNew[MAX_THREADS+2];
	__shared__ int BB_S[4][MAX_THREADS];
     //   memset(matched, 0, n*sizeof(int));
	int w=threadIdx.x+1;

	for(int i=0;i<4;i++)
	{
		BB_S[i][threadIdx.x]= BB_d[i*J+threadIdx.x];
	}

	{
		D[threadIdx.x] = 0;
		{
			D[w] = (1<<(k+1)) -1;

			for(int i = 0; i < lc - 1; i++)
			{
				D[w] = (D[w] << k+2) + (1<<(k+1)) -1;
			}
		}
		D[J+1] = (1<<((k+2)*lc)) - 1;
	}
	int startblock=(blockIdx.x == 0?start_addr:(start_addr+(blockIdx.x * (textBlockSize-overlap))));
//	int startblock=start_addr;
 //       int endBlock = (((startblock + textBlockSize) > n )? ((startblock +(n- (startblock)))-1):((startblock + textBlockSize)-1));
        int size= (((startblock + textBlockSize) > n )? ((n- (startblock))):( textBlockSize));

	int copyBlock=(size/J)+ ((size%J)==0?0:1);
//	int text_start_pos=startblock+(threadIdx.x*copyBlock);
	if((threadIdx.x * copyBlock) <= size)
	memcpy(Text_S+(threadIdx.x*copyBlock),text_d+(startblock+threadIdx.x*copyBlock),(((((threadIdx.x*copyBlock))+copyBlock) > size)?(size-(threadIdx.x*copyBlock)):copyBlock));



   // for(w=1; w < J + 2; w++) printf("D[%d] intitialized to %x\n", w, D[w]);
 //   uint32_t* Dnew = malloc((J+2)*32);*
    memcpy(DNew, D, (J+2)*sizeof(int));
    __syncthreads();
    uint32_t initial = D[1];
    uint32_t x;
    uint32_t mask = 1;
    for(int i = 0; i < lc - 1; i++)mask = (mask<<(k+2)) + 1;
//    printf("mask: %x\n", mask);
    for(int i = 0; i < size;i++)
    {
  //      printf("Char read: %c\n\n",  text[i]);
  //      for(w=1;w<=J;w++)
        {
            x =  ((D[w] >> (k+2)) | (D[w - 1] << ((k + 2)* (lc - 1))) | (BB_S[(((int)Text_S[i])/2)%4][w-1])) & ((1 << (k + 2)* lc) - 1);
            DNew[w] = ((D[w]<<1) | mask)
	     	    & (((D[w] << k+3) | mask|((D[w +1] >>((k+2)*(lc - 1)))<<1)))
                    & (((x + mask) ^ x) >> 1)
                    & initial;
        }
	__syncthreads();
        memcpy(D, DNew, (J+2)*sizeof(int));
	if(!(D[J] & 1<<(k + (k + 2)*(lc*J -m + k ))))
	{
		matched[startblock+i] = 1;
		//     D[J] |= (1<<(k+2) - 1);
		D[J] |= ((1<<(k + 1 + (k + 2)*(lc*J -m + k ))) - 1);
	}
    }

}
int main(void)
{
	hipEvent_t start, stop,stopAfterMatch;
	float time1,time2,time3;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&stopAfterMatch);


	hipEventRecord(start, 0);
	const char pattern_h[] = "TACACGAGGAGAGGAGAAGAACA";
	//const char pattern_h[] = "ACGACG";
	//const char text_h[] = "TTTACGGCG";
	//        const char text_h[] = "ACGATCGTAGCTAGTCGATGCTAGCTAGCTGATCGTACGTAGCTGTACGTAGCTATCGTAGCTACTGATCGTAGCTAGCTAGCGTAGTATATATTATACGTA";
	char * text_h=getTextString();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
	
	int k = 13;
	int i;
	int count=0;
	char *pattern_d, *text_d;     // pointers to device memory
	char* text_new_d;
	int* matched_d;
	int* matched_new_d;
	uint32_t* BB_d;
	uint32_t* BB_new_d;
	int* matched_h = (int*)malloc(sizeof(int)* strlen(text_h));
	hipMalloc((void **) &pattern_d, sizeof(char)*strlen(pattern_h)+1);
	hipMalloc((void **) &text_d, sizeof(char)*strlen(text_h)+1);
	hipMalloc((void **) &matched_d, sizeof(int)*strlen(text_h));
	hipMemcpy(pattern_d, pattern_h, sizeof(char)*strlen(pattern_h)+1, hipMemcpyHostToDevice);
	hipMemcpy(text_d, text_h, sizeof(char)*strlen(text_h)+1, hipMemcpyHostToDevice);
	hipMemset(matched_d, 0,sizeof(int)*strlen(text_h));

	int m = strlen(pattern_h);
	int n = strlen(text_h);
	if(k>= m)
	{
		printf("Error: Distance must be less than m\n");
		exit(0);
	}
	char revpatt[m];
	int rev=0,rev_start_pos=0;
	for(rev=0;rev<strlen(pattern_h);rev++)
		revpatt[rev]=pattern_h[strlen(pattern_h)-1-rev];
	revpatt[rev]='\0';

	uint32_t* BB_h[4];
        unsigned int maxLc = ((((m-k)*(k+2)) > (31))?(31/(k+2)):(m-k));
	unsigned int lc=2;
	if(lc>maxLc)
	{
		printf("Error: Maximum Diagonals possible is %d but you entered %d\n",maxLc,lc);
		exit(0);
	}
	unsigned int noWordorNfa =((m-k)/lc) + (((m-k)%lc)  == 0?0:1);
	hipMalloc((void **) &BB_d, sizeof(int)*noWordorNfa*4);
	printf("# of diagonals packed into a word: %d, number of automata/words: %d\n , m %d , k %d  n %d \n", lc,noWordorNfa,m,k,n);
	if(noWordorNfa >= MAX_THREADS)
	{
		printf("Error: max threads\n");
		exit(0);
	}

	calculateBBArray(BB_h,pattern_h,m,k,lc,noWordorNfa);

	for(i=0;i<4;i++)
	{
		hipMemcpy(BB_d+ i*noWordorNfa, BB_h[i], sizeof(int)*noWordorNfa, hipMemcpyHostToDevice);
	}
	int overlap=m;
	int textBlockSize=(((m+k+1)>n)?n:(m+k+1));
	hipStream_t stream[MAX_STREAMS];
	for(i=0;i<MAX_STREAMS;i++)
		hipStreamCreate( &stream[i] );
//	int maxNoBlocks=((1 + ((n-textBlockSize)/(textBlockSize-overlap)) + (((n-textBlockSize)%(textBlockSize-overlap)) == 0?0:1)));
	int start_addr=0,index=0,maxNoBlocks=0;
	if(textBlockSize>n)
	{
		maxNoBlocks=1;
	}
	else
	{
		 maxNoBlocks=((1 + ((n-textBlockSize)/(textBlockSize-overlap)) + (((n-textBlockSize)%(textBlockSize-overlap)) == 0?0:1)));
	}
	int kernelBlocks = ((maxNoBlocks > MAX_BLOCKS)?MAX_BLOCKS:maxNoBlocks);
 	int blocksRemaining =maxNoBlocks;
	printf(" maxNoBlocks %d kernel Blocks %d \n",maxNoBlocks,kernelBlocks);
	while(blocksRemaining >0)
	{
	kernelBlocks = ((blocksRemaining > MAX_BLOCKS)?MAX_BLOCKS:blocksRemaining);
	printf(" Calling %d Blocks with starting Address %d , textBlockSize %d \n",kernelBlocks,start_addr,textBlockSize);
	match<<<kernelBlocks,noWordorNfa,0,stream[(index++)%MAX_STREAMS]>>>(BB_d,text_d,n,m,k,noWordorNfa,lc,start_addr,textBlockSize,overlap,matched_d);
	start_addr+=kernelBlocks*(textBlockSize-overlap);;
	blocksRemaining -= kernelBlocks;
	}
	hipMemcpy(matched_h, matched_d, sizeof(int)*strlen(text_h), hipMemcpyDeviceToHost);
	checkCUDAError("Matched Function");
	for(i=0;i<MAX_STREAMS;i++)
		hipStreamSynchronize( stream[i] );	
	for(int i=0; i<strlen(text_h); i++)
	{
		if(matched_h[i] == 1){

			rev_start_pos=((i-(m+k)<0)?0:(i-(m+k)));
			char revtext[i-rev_start_pos+1];
			//printf(" size %d %d", startpos, (i-startpos));
			for(rev=0;rev<(i-rev_start_pos);rev++)
				revtext[rev]=text_h[i-rev];
			revtext[rev]='\0';
			int* matched_new_h = (int*)malloc(sizeof(int)* (i-rev_start_pos));
			//matchFn(revtext,revpatt,k,matchednew);
			uint32_t* BB_new_h[4];
			calculateBBArray(BB_new_h,revpatt,m,k,lc,noWordorNfa);
			hipMalloc((void **) &BB_new_d, sizeof(int)*noWordorNfa*4);
			for(int l=0;l<4;l++)
			{
				hipMemcpy(BB_new_d+ l*noWordorNfa, BB_new_h[l], sizeof(int)*noWordorNfa, hipMemcpyHostToDevice);
			}
			hipMalloc((void **) &text_new_d, sizeof(char)*(i-rev_start_pos+1));
			hipMalloc((void **) &matched_new_d, sizeof(int)*(i-rev_start_pos));
			hipMemcpy(text_new_d, revtext, sizeof(char)*(i-rev_start_pos+1), hipMemcpyHostToDevice);
			hipMemset(matched_new_d, 0,sizeof(int)*(i-rev_start_pos));

			match<<<1,noWordorNfa>>>(BB_new_d,text_new_d,i-rev_start_pos,m,k,noWordorNfa,lc,0,i-rev_start_pos ,0,matched_new_d);
			hipMemcpy(matched_new_h, matched_new_d, sizeof(int)*(i-rev_start_pos), hipMemcpyDeviceToHost);
			for(rev=0;rev<(i-rev_start_pos);rev++){
				if(matched_new_h[rev]== 1)
				{
					// printf(" startpos values %d \n ",rev);
					break; }
			}
			count++;printf("match from : start %d to end %d\n", i-rev, i); }
	}   
	printf("Total matches is %d\n",count);
	hipEventRecord(stopAfterMatch, 0);

	hipEventSynchronize(stopAfterMatch);
	hipEventElapsedTime(&time2, start, stopAfterMatch);
	hipEventElapsedTime(&time3, stop, stopAfterMatch);
	printf ("Time for the kernel: %f ms for file Op and Total time taken is %f ms, Match time is %f ms  \n", time1,time2,time3);


	free(matched_h);
	hipFree(pattern_d);hipFree(text_d);hipFree(matched_d);

}

void calculateBBArray(uint32_t** BB,const char* pattern,int m,int k , int lc,int J)
{

	uint32_t* B[4];
	int i,w;
	uint32_t mask;
	  
	for(i=0; i<4; i++)
	{
		B[i] = (uint32_t*)malloc(J*sizeof(uint32_t));
		for(w = 0; w < J;w++) {
			B[i][w] = (1<<(k+lc)) - 1;
		}
	}

	i = 0;
	int iter = 0;
	while(i < m)
	{
		for(w = 0;w < k + lc;w++) {
			if(iter < J) {
				if(i + w < m) {
					 mask = ~(1 << w);
					if(pattern[i + w] == 'A')B[0][iter] &= mask;
					if(pattern[i + w] == 'C')B[1][iter] &= mask;
					if(pattern[i + w] == 'T')B[2][iter] &= mask;
					if(pattern[i + w] == 'G')B[3][iter] &= mask;
				}
				else {
					int cnt;
					for(cnt = 0; cnt < 4;cnt++) B[cnt][iter] &= ~(1 << w);
				}
			}
		}
		i += w - k;
		++iter;
	}
/*	for(i = 0;i < 4;i++) {
		//printf("C[%d] is now : ", i);
		for(int w = 0; w < J;w++) {
			printf("%x,\t", B[i][w]);
		}
		printf("\n");
	}
*/

	mask = (1<<(k+1)) - 1;

	for(i = 0; i < 4; i++)
	{
		int j,w;
		int shift;
		BB[i] = (uint32_t* )malloc(J * sizeof(int));
		memset(BB[i], 0, J*sizeof(int));
		for(w =0; w<J; w++)
		{
			for(j = 0; j < lc; j++)
			{
				if((w*lc + j) >= (m-k)) continue;
				shift = (lc-j-1)*(k+2);
				BB[i][w] |= ((((B[i][w]>> j))& mask) << shift);
				//printf("chunk: %x, shifted by %d\n", (B[i]>>j)& mask, shift);
				//printf("iteration %d:  OR'ed with %x,\n BB is now %x\n", j,((B[i]>>j)& mask)<<(m-k-j-1)*(k+2), BB[i]);

			}
			//printf("BB[%d][%d]: %x \t",i, w, BB[i][w]);

		}
	//	printf("\n");
	}

}

